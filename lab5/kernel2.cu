#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define THREAD_W 32
#define THREAD_H 32
#define BLOCK_W 50
#define BLOCK_H 38
using namespace std;

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}
//float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations, stepX, stepY
__global__ void mandelKernel(
    int *ans, float lowerX, float lowerY,
    int resX, int resY, int maxIterations, float stepX, float stepY) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    // 根據 CUDA 模型，算出當下 thread 對應的 x 與 y
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    long long int idx = j * resX + i;
    float x = lowerX + i * stepX;
    float y = lowerY + j * stepY;
    ans[idx] = mandel(x, y, maxIterations);
    // printf("blockinx.x = %d, blockdim.x = %d, threadidx.x = %d\n", blockIdx.x ,blockDim.x , threadIdx.x);
    // printf("blockinx.y = %d, blockdim.y = %d, threadidx.y = %d\n", blockIdx.y ,blockDim.y , threadIdx.y);
    // printf("i = %d, j = %d, x = %f, y = %f, ans[%d] = %d\n", i,j,x,y,idx, ans[idx]);
}

// Host front-end function that allocates the memory and launches the GPU kernel
// hostFE(x1, y1, x0, y0, output, width = 1600, height = 1200, maxIterations);
void hostFE (
    float upperX, float upperY, float lowerX, float lowerY, 
    int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    dim3 blockSize(BLOCK_W, BLOCK_H);
    dim3 numBlock(THREAD_W, THREAD_H);
    int *ans_h, *ans_c;
    // cudaHostAlloc( (void**)&ans_h, resX * resY * sizeof( *ans_h ),cudaHostAllocDefault );
    hipHostAlloc( (void**)&ans_h, resX * resY * sizeof(int),hipHostMallocDefault );
    for(int i = 0; i<resX*resY; i++)
      ans_h[i] = 0;
    size_t pitch;
    // 宣告 Device (GPU) 記憶體
    hipMallocPitch((void **)&ans_c, &pitch, resX * sizeof(int), resY);
    mandelKernel<<<blockSize, numBlock>>>(ans_c, lowerX, lowerY, resX, resY, maxIterations, stepX, stepY);

    // 等待 GPU 所有 thread 完成
    hipDeviceSynchronize();
    
    // 將 Device 的資料傳回給 Host
    hipMemcpy(img, ans_c, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    // for(int i = 0; i < (resX * resY); i++)
    //     cout<<img[i]<<" ";
    

    hipFree(ans_c);
    /*
    cout<<"width = "<<resX<<" height= "<<resY<<endl;
    int dev = 0;
    cudaDeviceProp deviceProp;
    cudaGetDeviceProperties(&deviceProp, dev);
    unsigned int maxThreads = deviceProp.maxThreadsPerBlock; //1024
    int *maxGrid = deviceProp.maxGridSize;
    cout<<maxGrid[0]<<endl;
    */
}
